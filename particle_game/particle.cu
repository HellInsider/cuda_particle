#include "hip/hip_runtime.h"
#include "particle.cuh"
#include "physics.cuh"
#include "math.h"
#include <thrust/device_vector.h>

#include "cudaGL.h" // for kernel function surf2Dwrite
#include "" // for kernel vars blockIdx and etc.

#define min(a, b) (a > b)? b: a
#define max(a, b) (a > b)? a: b 
//__device__ __constant__ physics_manager phManager;


__global__ void Fill(hipSurfaceObject_t s, dim3 texDim)
{
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= texDim.x || y >= texDim.y)
  {
    return;
  }

//  uchar4 data = make_uchar4(255.0f * x / texDim.x, 255.0f * y / texDim.y, 0x00, 0xff);
  uchar4 data = make_uchar4(0x88, 0xBB, 0xBB, 0xff);
  surf2Dwrite(data, s, x * sizeof(uchar4), y);
}

__global__ void DrawShapes(hipSurfaceObject_t s)
{
  unsigned int i = blockIdx.x;
  if (i > shapesDevice.nShapes)
    return;
  shape shp = shapesDevice.shapes[i];
}

__global__ void DrawParticles(hipSurfaceObject_t s, particle* poolCur, dim3 texDim)
{
  unsigned int i = blockIdx.x;
  particle part = poolCur[i];
  unsigned int x = part.x;
  unsigned int y = part.y;

  if (x + 1 >= texDim.x || y + 1 >= texDim.y || part.type == PART_DEAD)
  {
    return;
  }
  float r = 0, g = 0, b = 0;
  switch (part.type) 
  {
  case PART_FIRST: {r = 255; g = 255; break; }
  case PART_SECOND: {r = 128; g = 64; break; }
  case PART_THIRD: {g = 200; break; }

  }

  uchar4 data = make_uchar4(r, g, b, 0xff);
  surf2Dwrite(data, s, x * sizeof(uchar4), y);
  surf2Dwrite(data, s, (x + 1) * sizeof(uchar4), y);
  surf2Dwrite(data, s, x * sizeof(uchar4), y + 1);
  surf2Dwrite(data, s, (x + 1) * sizeof(uchar4), y + 1);
}

__device__ float dist(particle x, particle y) 
{
    //return float(sqrt(pow((x.x - y.x), 2) + pow((x.y - y.y), 2)));
    return float(fabs(x.x - y.x) + fabs(x.y - y.y)); //trying to accelerate app. Manhattan's metrics
}

__device__ void CollisionCheck(particle* poolPrev, particle* poolCur, int maxParticles)
{
    unsigned int i = blockIdx.x;
    float t;
    for (int j = i+1; j < maxParticles; j++)
    {
        if (poolPrev[i].type != PART_DEAD && poolPrev[j].type != PART_DEAD)
        {
            if (poolPrev[i].type != poolPrev[j].type)
            {
                if (dist(poolPrev[i], poolPrev[j]) < 2) 
                {
                    //first method
                    /*t = poolPrev[i].vx;
                    poolPrev[i].vx = 0.5 * (poolPrev[j].vx + t);
                    poolPrev[j].vx = 0.5 * (poolPrev[j].vx + t);

                    t = poolPrev[i].vy;
                    poolPrev[i].vy = 0.5 * (poolPrev[j].vy + t);
                    poolPrev[j].vy = 0.5 * (poolPrev[j].vy + t);*/

                    //second method
                    t = poolPrev[i].vx;
                    poolPrev[i].vx = poolPrev[j].vx;
                    poolPrev[j].vx = t;

                    t = poolPrev[i].vy;
                    poolPrev[i].vy = poolPrev[j].vy;
                    poolPrev[j].vy = t;

                    //third method
                    /*poolPrev[i].vx += poolPrev[j].vx;
                    poolPrev[i].vy += poolPrev[j].vy;
                    poolPrev[j].vx = 2 * poolPrev[j].vx + poolPrev[i].vx;
                    poolPrev[j].vy = 2 * poolPrev[j].vy + poolPrev[i].vy;*/

                }
            }
        }
        
    }
}

__global__ void Update(particle* poolPrev, particle* poolCur, double timeDelta, int maxParticles)
{
    unsigned int i = blockIdx.x;
    //phManager.physicsMakeAction(&poolCur[i]);
    if (poolCur[i].type == PART_DEAD)
      return;
    
    ShapesCollisionCheck(&poolCur[i], timeDelta);
    poolCur[i].vy -= 0.00015 * timeDelta;   // IDD: I will remove const soon. After physics discussion
    poolCur[i].x = poolPrev[i].x + poolPrev[i].vx * timeDelta;
    poolCur[i].y = poolPrev[i].y + poolPrev[i].vy * timeDelta;
    
    CollisionCheck(poolPrev, poolCur, maxParticles);
    poolCur[i].remainingAliveTime = max(poolPrev[i].remainingAliveTime - timeDelta, 0.f);
    poolCur[i].type = poolPrev[i].remainingAliveTime > 0 ? poolPrev[i].type : PART_DEAD;
}

__device__ unsigned seed = 123456789;
__device__ unsigned random(void)
{
  unsigned a = 1103515245;
  unsigned c = 12345;
  unsigned m = 1 << 31;
  seed = (a * seed + c) % m;
  return seed;
}
__global__ void Spawn(particle* poolCur, int maxParticles)
{
  int startSlot = 0;
  for (int i = 0; i < spawnersDevice.nSpawners; i++)
  {
    spawner sp = spawnersDevice.spawners[i];
    int numToSpawn = sp.intensity;
    for (int j = 0; j < numToSpawn; j++)
      for (int k = startSlot; k < maxParticles; k++) // max particles here
        if (poolCur[k].type == PART_DEAD)
        {

          particle p = { sp.x, sp.y, sp.vx + (random() % sp.directionsCount) * sp.spread, sp.vy + (random() % sp.directionsCount) * sp.spread, 
                            sp.type, sp.particleAliveTime, sp.particleAliveTime, sp.phType };

          poolCur[k] = p;
          startSlot = k + 1;
          break;
        }
  }
}

void part_mgr::Compute(hipSurfaceObject_t s, dim3 texSize, double timeDelta)
{
  hipMemcpyToSymbol(HIP_SYMBOL(shapesDevice), &shapesHost, sizeof(shapes_cbuf));

  dim3 thread(1);
  dim3 block(MAX_PARTICLES);
  dim3 oneBlock(1);

  Spawn<<< oneBlock, thread >>>(partPoolCur, MAX_PARTICLES);
  Update<<< block, thread >>>(partPoolCur, partPoolCur, timeDelta, MAX_PARTICLES);
  DrawParticles <<< block, thread >>>(s, partPoolCur, texSize);

}

void part_mgr::Init(void)
{
    hipError_t cudaStatus = hipSuccess;
    particle tmp[MAX_PARTICLES];
    for (int i = 0; i < MAX_PARTICLES; i++)
    {
        particle p = { 0, 0, 0, 0, PART_DEAD, 0, 0, SPACE };
        tmp[i] = p;
    }
    cudaStatus = hipMalloc(&partPoolCur, sizeof(particle) * MAX_PARTICLES);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "failed!");
    //cudaStatus = hipMalloc(&partPoolCur, sizeof(particle) * MAX_PARTICLES);
    //if (cudaStatus != hipSuccess)
    //    fprintf(stderr, "failed!");

    hipMemcpy(partPoolCur, tmp, sizeof(particle) * MAX_PARTICLES, hipMemcpyHostToDevice);
    //hipMemcpy(partPoolCur, tmp, sizeof(particle) * MAX_PARTICLES, hipMemcpyHostToDevice);

    spawnersHost.nSpawners = 3;
    spawnersHost.spawners[0] = { 600, 500, 0.35, 0.35, PART_FIRST, 0.005, 1, 8, 3000, EARTH_PHYSICS };
    spawnersHost.spawners[1] = {700, 700, 0.25, -0.15, PART_SECOND, -0.008, 2, 10, 3000, SPACE};
    spawnersHost.spawners[2] = {1000, 300, -0.15, 0.45, PART_THIRD, -0.005, 1, 10, 1500, SPACE};
    hipMemcpyToSymbol(HIP_SYMBOL(spawnersDevice), &spawnersHost, sizeof(spawner_cbuf));
}

void part_mgr::Kill(void)
{
  hipError_t cudaStatus = hipSuccess;

  cudaStatus = hipFree(partPoolCur);
  if (cudaStatus != hipSuccess)
    fprintf(stderr, "failed!");
  //cudaStatus = hipFree(partPoolCur);
  //if (cudaStatus != hipSuccess)
   // fprintf(stderr, "failed!");

}

void part_mgr::AddCircle(float cx, float cy, float radius)
{
  if (shapesHost.nShapes == MAX_SHAPES)
    return;
  shapesHost.shapes[shapesHost.nShapes] = { SHAPE_CIRCLE, cx, cy, radius, 0 };
  shapesHost.nShapes++;
}

void part_mgr::AddSquare(float x1, float y1, float x2, float y2)
{
  if (shapesHost.nShapes == MAX_SHAPES)
    return;
  shapesHost.shapes[shapesHost.nShapes] = { SHAPE_SQUARE, x1, y1, x2, y2 };
  shapesHost.nShapes++;
}

void part_mgr::AddSegment(float x1, float y1, float x2, float y2)
{
  if (shapesHost.nShapes == MAX_SHAPES)
    return;
  shapesHost.shapes[shapesHost.nShapes] = { SHAPE_SEGMENT, x1, y1, x2, y2 };
  shapesHost.nShapes++;
}

const shapes_cbuf& part_mgr::GetShapes(void)
{
  return shapesHost;
}


struct pt {
    float x, y;
};

__device__ void ShapesCollisionCheck(particle* part, double timeDelta)
{

    float shiftX, shiftY;
    shiftX = part->vx * timeDelta;
    shiftY = part->vy * timeDelta;
    shape sh;

    for (int i = 0; i < shapesDevice.nShapes; i++)
    {
        sh = shapesDevice.shapes[i];
        switch (sh.type)
        {
        case SHAPE_SQUARE: {SquareCollision(&sh, part, shiftX, shiftY); break; }
        case SHAPE_SEGMENT: {SegmentCollision(&sh, part, shiftX, shiftY); break; }
        case SHAPE_CIRCLE: {CircleCollision(&sh, part, shiftX, shiftY); break; }
        }
    }
}

__device__ void CircleCollision(shape* shape, particle* part, float shiftX, float shiftY)
{
    if (pow(part->y + shiftY - shape->params[1], 2) + pow(part->x + shiftX - shape->params[0], 2) <= shape->params[2] * shape->params[2])
    {
        pt prtcl = { part->vx, part->vy };
        pt norm = { shape->params[0] - part->x, shape->params[1] - part->y };
        float len = sqrt(pow(norm.x, 2) + pow(norm.y, 2));
        norm = { norm.x / len, norm.y / len };
        float t = 2 * (prtcl.x * norm.x + prtcl.y * norm.y);
        norm = { t * norm.x, t * norm.y };
        pt res = { prtcl.x - norm.x, prtcl.y - norm.y };
        part->vx = res.x;
        part->vy = res.y;
    }  
}

__device__ void SquareCollision(shape* shape, particle* part, float shiftX, float shiftY)
{
    float newX = part->vx, newY = part->vy;
    if (part->x + shiftX <= shape->params[0] && part->x + shiftX >= shape->params[2]
        && part->y <= shape->params[1] && part->y >= shape->params[3])
    {
        newX *= -0.8;   //slowdown after hit
    }

    if (part->y + shiftY <= shape->params[1] && part->y + shiftY >= shape->params[3]
        && part->x <= shape->params[0] && part->x >= shape->params[2])
    {
        newY *= -0.8;    //slowdown after hit
    }

    part->vx = newX;
    part->vy = newY;
}


__device__  int area(pt a, pt b, pt c) {
    return (b.x - a.x) * (c.y - a.y) - (b.y - a.y) * (c.x - a.x);
}

__device__  bool intersect_1(float a, float b, float c, float d) {
    float t;
    if (a > b)
    {
        t = a;
        a = b;
        b = t;
    }
    if (c > d)
    {
        t = c;
        c = d;
        d = t;
    }
    return max(a, c) <= min(b, d);
}

__device__ int sign(float x)
{
    float eps = 1e-1;
    return x > eps ? 1 : (x < eps ? -1 : 0);
}

__device__ bool intersect(pt a, pt b, pt c, pt d) 
{

    return intersect_1(a.x, b.x, c.x, d.x)
        && intersect_1(a.y, b.y, c.y, d.y)
        && sign(area(a, b, c)) * sign(area(a, b, d)) <= 0
        && sign(area(c, d, a)) * sign(area(c, d, b)) <= 0;
}


__device__ void SegmentCollision(shape* shape, particle* part, float shiftX, float shiftY)
{
    if (intersect(pt{ shape->params[0], shape->params[1] }, pt{ shape->params[2], shape->params[3] },
        pt{ part->x, part->y }, pt{ part->x + shiftX, part->y + shiftY }))
    {
        pt prtcl = { part->vx, part->vy };
        pt norm = { shape->params[1] - shape->params[3], - shape->params[0] + shape->params[2] };
        float side = sign(area(pt{ shape->params[0], shape->params[1] },
            pt{ shape->params[2], shape->params[3] }, pt{ part->x, part->y }));
        norm = { norm.x * side, norm.y * side };

        float len = sqrt(pow(norm.x, 2) + pow(norm.y, 2));
        norm = { norm.x / len, norm.y / len };
        float t = 2 * (prtcl.x * norm.x + prtcl.y * norm.y);
        norm = { t * norm.x, t * norm.y };
        pt res = { prtcl.x - norm.x, prtcl.y - norm.y };
        part->vx = res.x;
        part->vy = res.y;
    }
}




