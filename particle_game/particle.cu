#include "hip/hip_runtime.h"
#include "particle.h"
#include "physics.h"
#include "math.h"

#include "cudaGL.h" // for kernel function surf2Dwrite
#include "" // for kernel vars blockIdx and etc.

__device__ __constant__ spawner_cbuf spawnersDevice;
__device__ __constant__ shapes_cbuf shapesDevice;
//__device__ __constant__ physics_manager phManager;


__global__ void Fill(hipSurfaceObject_t s, dim3 texDim)
{
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= texDim.x || y >= texDim.y)
  {
    return;
  }

//  uchar4 data = make_uchar4(255.0f * x / texDim.x, 255.0f * y / texDim.y, 0x00, 0xff);
  uchar4 data = make_uchar4(0x88, 0xBB, 0xBB, 0xff);
  surf2Dwrite(data, s, x * sizeof(uchar4), y);
}

__global__ void DrawShapes(hipSurfaceObject_t s)
{
  unsigned int i = blockIdx.x;
  if (i > shapesDevice.nShapes)
    return;
  shape shp = shapesDevice.shapes[i];
}

__global__ void DrawParticles(hipSurfaceObject_t s, particle* poolCur, dim3 texDim)
{
  unsigned int i = blockIdx.x;
  particle part = poolCur[i];
  unsigned int x = part.x;
  unsigned int y = part.y;

  if (x + 1 >= texDim.x || y + 1 >= texDim.y || part.type == PART_DEAD)
  {
    return;
  }
  float r = part.type == PART_FIRST ? 255 : 128;
  float g = part.type == PART_FIRST ? 255 : 64;
  float b = 0;
  uchar4 data = make_uchar4(r, g, b, 0xff);
  surf2Dwrite(data, s, x * sizeof(uchar4), y);
  surf2Dwrite(data, s, (x + 1) * sizeof(uchar4), y);
  surf2Dwrite(data, s, x * sizeof(uchar4), y + 1);
  surf2Dwrite(data, s, (x + 1) * sizeof(uchar4), y + 1);
}

__device__ float dist(particle x, particle y) 
{
    //return float(sqrt(pow((x.x - y.x), 2) + pow((x.y - y.y), 2)));
    return float(fabs(x.x - y.x) + fabs(x.y - y.y)); //trying to accelerate app. Manhattan's metrics
}

__device__ void CollisionCheck(particle* poolPrev, particle* poolCur, int maxParticles)
{
    unsigned int i = blockIdx.x;
    float t;
    for (int j = i+1; j < maxParticles; j++)
    {
        if (poolPrev[i].type != PART_DEAD && poolPrev[j].type != PART_DEAD)
        {
            if (poolPrev[i].type != poolPrev[j].type)
            {
                if (dist(poolPrev[i], poolPrev[j]) < 2) 
                {
                    //first method
                    /*t = poolPrev[i].vx;
                    poolPrev[i].vx = 0.5 * (poolPrev[j].vx + t);
                    poolPrev[j].vx = 0.5 * (poolPrev[j].vx + t);

                    t = poolPrev[i].vy;
                    poolPrev[i].vy = 0.5 * (poolPrev[j].vy + t);
                    poolPrev[j].vy = 0.5 * (poolPrev[j].vy + t);*/

                    //second method
                    t = poolPrev[i].vx;
                    poolPrev[i].vx = poolPrev[j].vx;
                    poolPrev[j].vx = t;

                    t = poolPrev[i].vy;
                    poolPrev[i].vy = poolPrev[j].vy;
                    poolPrev[j].vy = t;

                    //third method
                    /*poolPrev[i].vx += poolPrev[j].vx;
                    poolPrev[i].vy += poolPrev[j].vy;
                    poolPrev[j].vx = 2 * poolPrev[j].vx + poolPrev[i].vx;
                    poolPrev[j].vy = 2 * poolPrev[j].vy + poolPrev[i].vy;*/

                }
            }
        }
        
    }
}

__global__ void Update(particle* poolPrev, particle* poolCur, double timeDelta, int maxParticles)
{
    unsigned int i = blockIdx.x;
    //phManager.physicsMakeAction(&poolCur[i]);
    if (poolCur[i].type == PART_DEAD)
      return;
    poolCur[i].vy -= 0.00015 * timeDelta;   //����� ��������� �����, ����� ������� ������ � �������
    poolCur[i].x = poolPrev[i].x + poolPrev[i].vx * timeDelta;
    poolCur[i].y = poolPrev[i].y + poolPrev[i].vy * timeDelta;
    CollisionCheck(poolPrev, poolCur, maxParticles);
    poolCur[i].remainingAliveTime = max(poolPrev[i].remainingAliveTime - timeDelta, 0.f);
    poolCur[i].type = poolPrev[i].remainingAliveTime > 0 ? poolPrev[i].type : PART_DEAD;
}

__device__ unsigned seed = 123456789;
__device__ unsigned random(void)
{
  unsigned a = 1103515245;
  unsigned c = 12345;
  unsigned m = 1 << 31;
  seed = (a * seed + c) % m;
  return seed;
}
__global__ void Spawn(particle* poolCur, int maxParticles)
{
  int startSlot = 0;
  for (int i = 0; i < spawnersDevice.nSpawners; i++)
  {
    spawner sp = spawnersDevice.spawners[i];
    int numToSpawn = sp.intensity;
    for (int j = 0; j < numToSpawn; j++)
      for (int k = startSlot; k < maxParticles; k++) // max particles here
        if (poolCur[k].type == PART_DEAD)
        {

          particle p = { sp.x, sp.y, sp.vx + (random() % sp.directionsCount) * sp.spread, sp.vy + (random() % sp.directionsCount) * sp.spread, 
                            sp.type, sp.particleAliveTime, sp.particleAliveTime, sp.phType };

          poolCur[k] = p;
          startSlot = k + 1;
          break;
        }
  }
}

void part_mgr::Compute(hipSurfaceObject_t s, dim3 texSize, double timeDelta)
{
  hipMemcpyToSymbol(HIP_SYMBOL(shapesDevice), &shapesHost, sizeof(shapes_cbuf));

  dim3 thread(1);
  dim3 block(MAX_PARTICLES);
  dim3 oneBlock(1);

  Spawn<<< oneBlock, thread >>>(partPoolCur, MAX_PARTICLES);
  Update<<< block, thread >>>(partPoolCur, partPoolCur, timeDelta, MAX_PARTICLES);
  DrawParticles <<< block, thread >>>(s, partPoolCur, texSize);

}

void part_mgr::Init(void)
{
    hipError_t cudaStatus = hipSuccess;
    particle tmp[MAX_PARTICLES];
    for (int i = 0; i < MAX_PARTICLES; i++)
    {
        particle p = { 0, 0, 0, 0, PART_DEAD, 0, 0, SPACE };
        tmp[i] = p;
    }
    cudaStatus = hipMalloc(&partPoolCur, sizeof(particle) * MAX_PARTICLES);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "failed!");
    //cudaStatus = hipMalloc(&partPoolCur, sizeof(particle) * MAX_PARTICLES);
    //if (cudaStatus != hipSuccess)
    //    fprintf(stderr, "failed!");

    hipMemcpy(partPoolCur, tmp, sizeof(particle) * MAX_PARTICLES, hipMemcpyHostToDevice);
    //hipMemcpy(partPoolCur, tmp, sizeof(particle) * MAX_PARTICLES, hipMemcpyHostToDevice);

    spawnersHost.nSpawners = 3;
    spawnersHost.spawners[0] = { 600, 500, -0.35, 0.35, PART_FIRST, 0.005, 1, 8, 3000, EARTH_PHYSICS };
    spawnersHost.spawners[1] = {700, 700, -0.25, -0.15, PART_SECOND, -0.008, 2, 10, 3000, SPACE};
    //spawnersHost.spawners[2] = {500, 500, -0.00015, -0.00015, PART_SECOND, -0.05, 3, 10, 1500, SPACE};
    hipMemcpyToSymbol(HIP_SYMBOL(spawnersDevice), &spawnersHost, sizeof(spawner_cbuf));
}

void part_mgr::Kill(void)
{
  hipError_t cudaStatus = hipSuccess;

  cudaStatus = hipFree(partPoolCur);
  if (cudaStatus != hipSuccess)
    fprintf(stderr, "failed!");
  //cudaStatus = hipFree(partPoolCur);
  //if (cudaStatus != hipSuccess)
   // fprintf(stderr, "failed!");

}

void part_mgr::AddCircle(float cx, float cy, float radius)
{
  if (shapesHost.nShapes == MAX_SHAPES)
    return;
  shapesHost.shapes[shapesHost.nShapes] = { SHAPE_CIRCLE, cx, cy, radius, 0 };
  shapesHost.nShapes++;
}

void part_mgr::AddSquare(float x1, float y1, float x2, float y2)
{

}

void part_mgr::AddSegment(float x1, float y1, float x2, float y2)
{

}



